#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>

int *compteur;

__global__ void k_incremente(int* valeur){
    if(valeur==5){
        return 0;
    }
    valeur+=1;
    k_incremente(valeur);
}

void init(){
    compteur=0;
}

int main(int argc, char **argv){
    hipMallocManaged(&compteur,sizeof(int));
    init()
    k_incremente<<<1,1>>>(compteur);
    hipDeviceSynchronize();

    printf("%d\n",compteur);

    hipFree(compteur);
    return 0;
}