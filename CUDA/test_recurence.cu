#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>

int *compteur;

__global__ void k_incremente(int* valeur){
    if(valeur[0]==5){
        return;
    }
    valeur+=1;
    k_incremente(valeur);
}

void init(){
    compteur=0;
}

int main(int argc, char **argv){
    hipMallocManaged(&compteur,sizeof(int));
    compteur=0;
    k_incremente<<<1,1>>>(compteur);
    hipDeviceSynchronize();

    printf("%d\n",compteur);

    hipFree(compteur);
    return 0;
}