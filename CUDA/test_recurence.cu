#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>

#include "kernel.cu"

int *compteur;

void init(){
    compteur=0;
}

int main(int argc, char **argv){
    hipMallocManaged(&compteur,sizeof(int));
    compteur=0;
    h_k_incremente<<<1,1>>>(compteur);
    hipDeviceSynchronize();

    printf("%d\n",compteur);

    hipFree(compteur);
    return 0;
}