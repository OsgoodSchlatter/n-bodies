#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>

//#include <helper_cuda.h>

#include<math.h>

#define NSTEP 1000

int const graphEtage=2;
int const n = pow(4,graphEtage);
int n_node=0;

int *valeur;

__global__ void k_set(int *valeur,int index,int lap){
    valeur[index]=lap;
}


void init(){
    for (int i=0;i<graphEtage;i++){
        n_node+=pow(4,i);
    }
    printf("n_node %d\n",n_node);


}

void recursiveLaunch(int index,int lap){
    if(index>n_node-n){
        return;
    }
    k_set<<<1,1>>>(valeur,index,lap);
    recursiveLaunch(index+1,lap);
}

int main(int argc, char **argv){
    init();
    hipMallocManaged(&valeur,n_node*sizeof(int));
    for (int i =0;i<n_node;i++){
        valeur[i]=-1;
    }

    bool graphCreated=false;
    hipGraph_t graph;
    hipGraphExec_t instance;
    hipStream_t stream;

    for(int istep=0; istep<NSTEP; istep++){
        if(!graphCreated){
            hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

            recursiveLaunch(*valeur,istep);

            hipStreamEndCapture(stream, &graph);
            hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
            graphCreated=true;
        }
        hipGraphLaunch(instance, stream);
        hipStreamSynchronize(stream);
    }

    hipFree(valeur);
    return 0;
}